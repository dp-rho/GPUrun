#include "hip/hip_runtime.h"
#include "commands.h"
#include "cuda_headers.h"

/* Memory for Rvar structures stored in __constant__ access memory for faster execution */
__constant__ Rvar gpu_vars[MAX_VARS];

/* Memory for Rvars that stored intermediate evaluations in __constant__ access memory */
__constant__ Rvar gpu_int_evals[MAX_INT_VARS];

/* Memory for size of loop iterations in __constant__ access memory for faster execution */
__constant__ int gpu_iter_lens[MAX_ITERS];

/* Memory for size of expressions stored in __constant__ access memory for faster execution */
__constant__ int gpu_evals_per_thread[MAX_EXPRS];

/* Memory to be allocated on the GPU that is only used in multi purpose intermediate  */
/* storage, the most common is intermediate storage after an expression evaluation    */
/* before a global variable is updated with the evaluated values                      */
double* scratch_gpu_memory;

/* Memory for linear algebra functions  */
double* gpu_Q;
double* gpu_tridiagonal;
double* gpu_eigvecs;
double* gpu_eigvals;


/* Define functions available to kernel */

/* Random number functions  */

/*
 * Generate a uniform random variable between a and b
 */

__device__ double runif_device(double a, double b, hiprandState_t* random_state) {
  double rs = hiprand_uniform_double(random_state);
  return ((b - a) * rs) + a;
}


/*
 * Uniform density
 */

__device__ double dunif_device(double a, double b) {
  return 1 / (b - a);
}


/*
 * Generate an exponential random variable with in scale
 * NOTE: Taken from rexp source code
 */

__device__ double rexp_device(double scale, hiprandState_t* random_state) {
  
  /* q[k-1] = sum(log(2)^k / k!)  k=1,..,n, */
  /* The highest n (here 16) is determined by q[n-1] = 1.0 */
  /* within standard precision */
  const static double q[] =
  {
    0.6931471805599453,
	  0.9333736875190459,
	  0.9888777961838675,
	  0.9984959252914960,
	  0.9998292811061389,
	  0.9999833164100727,
	  0.9999985691438767,
	  0.9999998906925558,
	  0.9999999924734159,
	  0.9999999995283275,
	  0.9999999999728814,
	  0.9999999999985598,
	  0.9999999999999289,
	  0.9999999999999968,
	  0.9999999999999999,
	  1.0000000000000000
  };

  double a = 0.;
  double u = hiprand_uniform_double(random_state);
  while(u <= 0. || u >= 1.) u = hiprand_uniform_double(random_state);
  for (;;) {
    u += u;
    if (u > 1.) break;
    a += q[0];
  }
  u -= 1.;

  if (u <= q[0]) return a + u;
    
  int i = 0;
  double ustar = hiprand_uniform_double(random_state); 
  double umin = ustar;
  do {
    ustar = hiprand_uniform_double(random_state);
    if (umin > ustar)
      umin = ustar;
    i++;
  } while (u > q[i]);

  return (a + umin * q[0]) * scale;
}


/*
 * Exponential density
 */

__device__ double dexp_device(double x, double scale) {
  return (1 / scale) * exp(-x / scale);
}

/*
 * Generate a normal random variable with mean mu and standard deviation sd
 */

__device__ double rnorm_device(double mean, double sd, hiprandState_t* random_state) {
  double rs = hiprand_normal_double(random_state);
  return (rs * sd) + mean;
}


/*
 * Normal density
 */

__device__ double dnorm_device(double x, double mean, double sd) {
  x = (x - mean) / sd;
  return M_1_SQRT_2PI * exp(-0.5 * x * x) / sd;
} 


/*
 * Generate a truncated normal variable with mean and standard deviation sd
 * NOTE: Implementation taken directly from Rtruncnorm package
 */


/* Exponential rejection sampling (a,inf) */
__device__ double ers_a_inf(double a, hiprandState_t* random_state) {
  const double ainv = 1.0 / a;
  double x, rho;
  do {
    x = rexp_device(ainv, random_state) + a;
    rho = exp(-0.5 * pow((x - a), 2));
  } while (runif_device(0, 1, random_state) > rho);
  return x;
}

/* Exponential rejection sampling (a,b) */
__device__ double ers_a_b(double a, double b, hiprandState_t* random_state) {
  const double ainv = 1.0 / a;
  double x, rho;
  do {
    x = rexp_device(ainv, random_state) + a;
    rho = exp(-0.5 * pow((x - a), 2));
  } while (runif_device(0, 1, random_state) > rho || x > b);
  return x;
}

/* Normal rejection sampling (a,b) */
__device__ double nrs_a_b(double a, double b, hiprandState_t* random_state) {
  double x = -DBL_MAX;
  while (x < a || x > b) {
    x = rnorm_device(0, 1, random_state);
  }
  return x;
}

/* Normal rejection sampling (a,inf) */
__device__ double nrs_a_inf(double a, hiprandState_t* random_state) {
  double x = -DBL_MAX;
  while (x < a) {
    x = rnorm_device(0, 1, random_state);
  }
  return x;
}

/* Half-normal rejection sampling */
__device__ double hnrs_a_b(double a, double b, hiprandState_t* random_state) {
  double x = a - 1.0;
  while (x < a || x > b) {
    x = rnorm_device(0, 1, random_state);
    x = fabs(x);
  }
  return x;
}

/* Uniform rejection sampling */
__device__ double urs_a_b(double a, double b, hiprandState_t* random_state) {
  
  const double phi_a = dnorm_device(a, 0, 1);
  double x = 0.0, u = 0.0;

  /* Upper bound of normal density on [a, b] */
  const double ub = a < 0 && b > 0 ? M_1_SQRT_2PI : phi_a;
  do {
    x = runif_device(a, b, random_state);
  } while (runif_device(0, 1, random_state) * ub > dnorm_device(x, 0, 1));
  return x;
}

/* Truncated on the left  */
__device__ double rtruncnorm_left(double a, double mean, double sd, 
                                  hiprandState_t* random_state) {
  const double alpha = (a - mean) / sd;
  if (alpha < T4) {
    return mean + sd * nrs_a_inf(alpha, random_state);
  } else {
    return mean + sd * ers_a_inf(alpha, random_state);
  }
}

/* Truncated on the right */
__device__ double rtruncnorm_right(double b, double mean, double sd, 
                                   hiprandState_t* random_state) {
  const double beta = (b - mean) / sd;
  return mean - sd * rtruncnorm_left(-beta, 0.0, 1.0, random_state);
}

/* General case */
__device__ double rtruncnorm_general(double a, double b, double mean, double sd,
                                     hiprandState_t* random_state) {
  const double alpha = (a - mean) / sd;
  const double beta = (b - mean) / sd;
  const double phi_a = dnorm_device(alpha, 0.0, 1.0);
  const double phi_b = dnorm_device(beta, 0.0, 1.0);
  if (alpha <= 0 && 0 <= beta) { 
    if (phi_a <= T2 || phi_b <= T1) {  
      return mean + sd * nrs_a_b(alpha, beta, random_state);
    } else { 
      return mean + sd * urs_a_b(alpha, beta, random_state);
    }
  } else if (alpha > 0) { 
    if (phi_a / phi_b <= T1) {
      return mean + sd * urs_a_b(alpha, beta, random_state);
    } else {
      if (alpha < T2) {
        return mean + sd * hnrs_a_b(alpha, beta, random_state);
      } else {
        return mean + sd * ers_a_b(alpha, beta, random_state);
      }
    }
  } else {
    if (phi_b / phi_a <= T2) {
      return mean - sd * urs_a_b(-beta, -alpha, random_state);
    } else {
      if (beta > -T3) {
        return mean - sd * hnrs_a_b(-beta, -alpha, random_state);
      } else {
        return mean - sd * ers_a_b(-beta, -alpha, random_state);
      }
    }
  }
  return 0;
}

/*
 * Generate a random truncated normal variable with potential truncation points a, b
 */

__device__ double rtruncnorm_device(double a, double b, double mean, double sd,
                                    hiprandState_t* random_state) {

  const int a_finite = (a == -DBL_MAX) ? 0 : 1;
  const int b_finite = (b == DBL_MAX) ? 0 : 1;

  if (a_finite && b_finite) {
    return rtruncnorm_general(a, b, mean, sd, random_state);
  } else if (!a_finite && b_finite) {
    return rtruncnorm_right(b, mean, sd, random_state);
  } else if (a_finite && !b_finite) {
    return rtruncnorm_left(a, mean, sd, random_state);
  } else if (!a_finite && !b_finite) {
    return rnorm_device(mean, sd, random_state);
  } 

  return 0;
}


/* Basic vector math functions  */

/*
 * Basic addition
 */

__host__ __device__ double add(double arg1, double arg2) {
  return arg1 + arg2;
}

/*
 * Basic subtraction
 */

__host__ __device__ double sub(double arg1, double arg2) {
  return arg1 - arg2;
}


/*
 * Basic multiplication
 */

__host__ __device__ double mul(double arg1, double arg2) {
  return arg1 * arg2;
}


/*
 * Basice division
 */

__host__ __device__ double dvs(double arg1, double arg2) {
  return arg1 / arg2;
}


/*
 * Range operator, i.e., from 1:10, return 1, 2, 3, ...
 */

__host__ __device__ double range(double arg1, double arg2, int data_index) {
  int sign = (arg2 > arg1) ? 1 : -1;
  int len = floor(abs(arg2 - arg1) + 1);
  return arg1 + (sign * (data_index % len));
}


/*
 * Matrix multiplication, implementation is naive with no use of shared memory
 */

__device__ double mat_mul(Rvar arg1, Rvar arg2, int data_index) {

  /* Check if evaluation index is out of bounds of return matrix  */
  if (data_index >= arg1.rdim * arg2.cdim) return 0;

  /* Identify the row and column index of the element being calculated  */
  int row_index = data_index % arg1.rdim;
  int col_index = data_index / arg1.rdim;

  /* The evaluated result of the input index  */
  double result = 0;

  /* Loop through the selected row and column of arg1 and arg2 and calculated dot product */
  for (int i = 0; i < arg1.cdim; i++) {
    result += (arg1.data[(i * arg1.rdim) + row_index] * 
               arg2.data[(col_index * arg2.rdim) + i]);
  }

  return result;
}


/*
 * Transpose matrix
 */

__device__ double transpose(Rvar arg, int data_index) {
 
  /* Check if evaluation index is out of bounds of return matrix  */
  if (data_index >= arg.rdim * arg.cdim) return 0;

  /* Identify the row and column index of the element being calculated  */
  int row_index = data_index % arg.cdim;
  int col_index = data_index / arg.cdim;

  /* Return the transposed index of the argument matrix */
  return arg.data[(arg.rdim * row_index) + col_index];
 
}


/*
 * Inverse matrix using Gauss-Jordan elimination
 * Note: This function does not return a value directly and instead
 * updates values in the pointer argument working_result
 */

__device__ void inverse(Rvar matrix_arg, double* working_copy, 
                        double* working_result,
                        int grid_index, int evals_per_thread, int grid_size, 
                        int thread_index, double* shared_mem_arr, 
                        cooperative_groups::grid_group grid) {

  /* Copy in matrix arg to the working copy, set working_result to identity matrix  */
  int data_index = grid_index;
  for (int i = 0; i < evals_per_thread; i++) {
    
    /* Check overflow */
    if (data_index >= matrix_arg.len) break;

    /* Copy data */
    working_copy[data_index] = matrix_arg.data[data_index];
    
    /* Fill in identity matrix  */
    if (data_index % matrix_arg.rdim == data_index / matrix_arg.rdim) {
      working_result[data_index] = 1;
    }
    else {
      working_result[data_index] = 0;
    }

    /* Update data index  */
    data_index += grid_size;
  }

  /* Sync grid before calculations begin  */
  grid.sync();

  /* Begin Gauss-Jordan elimination */
  for (int zero_col = 0; zero_col < matrix_arg.cdim; zero_col++) {
    int col_offset = matrix_arg.rdim * zero_col;    

    /* Ensure the diagnoal element of this column is not 0, if it is, ERROR  */
    /* TODO: diagonal elemenent */

    /* Sync all threads after row has been added  */
    grid.sync();

    /* Divide row of diagonal element by diagonal element */
    /* NOTE: Currently does not support matricies with more than SMs * 256 columns */
    double divisor = working_copy[(zero_col * matrix_arg.rdim) + zero_col];
    if (grid_index < matrix_arg.cdim) {
      working_copy[(grid_index * matrix_arg.rdim) + zero_col] /= divisor;
    }
    else if (grid_index < 2 * matrix_arg.cdim) {
      working_result[(grid_index - matrix_arg.cdim) * matrix_arg.rdim + zero_col] /= divisor;
    }
    grid.sync();

    /* Save the zeroing column in shared mem in each block  */
    /* NOTE: Not supported for matrices with more than 256 * MAX_EVALS_PER_THREAD rows  */
    data_index = thread_index;
    while (data_index < matrix_arg.rdim) {
      shared_mem_arr[data_index] = working_copy[(zero_col * matrix_arg.rdim) + data_index];
      data_index += THREADS_PER_BLOCK;
    }
    __syncthreads();
   
    /* Zero out col using Ri <- Ri - Rj x aij for all i =/= j, with j the zeroing col  */ 
    data_index = grid_index;
    for (int i = 0; i < evals_per_thread; i++) {
      
      /* Check overflow */
      if (data_index >= matrix_arg.len) break;

      int col_index = data_index / matrix_arg.rdim;
      int row_index = data_index % matrix_arg.rdim;

      if (row_index != zero_col) {
        working_copy[data_index] -= (working_copy[col_index * matrix_arg.rdim + zero_col] *
                                     shared_mem_arr[row_index]);
        working_result[data_index] -= (working_result[col_index * matrix_arg.rdim + zero_col] *
                                       shared_mem_arr[row_index]);
      }

      data_index += grid_size;
    }
    grid.sync();
  }
  
}


/*
 * Reduces symmetric PSD matrix argument to tridiagonal form and also stores Q,
 * with Q = P(n-2) %*% P(n-3) ... P(1) where P is the householder matrix used
 * to reduce input matrix at each iteration.
 * NOTE: No output is produced by this function, the global variables of 
 * gpu_Q and gpu_tridiagonal are updated.
 */

__device__ void householder_reduction(Rvar matrix_arg, double* gpu_Q,
                                      double* gpu_tridiagonal, double* shared_arr,  
                                      double* eval_memory, double* linalg_vec, int grid_size,
                                      int grid_index, int thread_index, int evals_per_thread,
                                      cooperative_groups::grid_group grid) {
  
  int row_index = grid_index % matrix_arg.rdim;
  int col_index = grid_index / matrix_arg.cdim;
  int data_index = grid_index;
  __shared__ double scaler1;
  __shared__ double scaler2;

  /* First initialize global memory for Q matrix and tridiagonal matrix */
  for (int i = 0; i < evals_per_thread; i++) {
    
    /* Check potential overflow */
    if (data_index > matrix_arg.len) break;

    /* Initialize matrix that will store tridiagonal results  */
    gpu_tridiagonal[data_index] = matrix_arg.data[data_index];

    /* Initialize Q to identity matrix */
    if (row_index == col_index) {
      gpu_Q[data_index] = 1;
    }
    else {
      gpu_Q[data_index] = 0;
    }

    data_index += grid_size;
    row_index = data_index % matrix_arg.rdim;
    col_index = data_index / matrix_arg.cdim;
  }

  /* Sync all threads after initialzation */
  grid.sync();

  /* Loop n-2 times to transform to tridiagonal form using householder matrices */
  for (int i = 0; i < (matrix_arg.rdim - 2); i++ ) {

    /* vectors x, u, p and q are all created in shared memory across all SMs  */
     
    /* create vector x as bottom n-(i+1) elements of current matrix */
    data_index = thread_index;
    while (data_index < matrix_arg.rdim) {
      if (data_index < (i + 1)) {
        shared_arr[data_index] = 0;
      }
      else {
        shared_arr[data_index] = gpu_tridiagonal[(i * matrix_arg.rdim) + data_index];
      }
      data_index += THREADS_PER_BLOCK;
    }
    __syncthreads();
    
    /* create vector u <- x + (sign) * norm(x) * e1, u overwrites x in shared_arr */
    /* This is not a parallel operation, as norm is sequential and only one index */
    /* is updated in the vector x, thus only the first thread of each block used  */
    if (thread_index == DEFAULT_DATA_INDEX) {
 
      /* determine sign for u = x + (sign) * e1 * norm(x) */
      int sign = (gpu_tridiagonal[matrix_arg.len - 1] > 0) ? 1 : -1;
 
      scaler1 = 0;
      for (int j = (i + 1); j < matrix_arg.rdim; j++) {
        scaler1 += (shared_arr[j] * shared_arr[j]);
      }

      /* Get norm of x  */
      double l1_norm = sqrt(scaler1);

      /* subtract the squared element that will be updated to create u  */
      scaler1 -= (shared_arr[i + 1] * shared_arr[i + 1]);

      /* update x to u, only element (i + 1) is updated  */
      shared_arr[i + 1] += (sign * l1_norm);

      /* get sum of squares for u, i.e., norm(u) ^ 2  */
      scaler1 += (shared_arr[i + 1] * shared_arr[i + 1]);
      
    }
    __syncthreads();

    /* create vector p <- A %*% u / (norm(u)^2 / 2) */
    data_index = thread_index;
    while (data_index < matrix_arg.rdim) {
      
      /* vector p is 0 for all indices < i  */
      if (data_index < i) {
        linalg_vec[data_index] = 0;
      }

      /* Calculate p[data_index] <- A[data_index,] %*% u / (norm(u)^2 / 2)  */
      else {

        /* initialize matrix multiplication result  */
        linalg_vec[data_index] = 0;

        /* vector u is 0 for index [0 - i], only need to multiply and sum (i + 1) onwards  */
        for (int j = (i + 1); j < matrix_arg.rdim; j++) {

          /* data_index is equiavalent to row_index for this loop */
          linalg_vec[data_index] += (gpu_tridiagonal[(j * matrix_arg.rdim) + data_index] * 
                                     shared_arr[j]);
        }

        /* Divide p[data_index] by H, with H <- norm(u)^2 / 2  */
        linalg_vec[data_index] /= (scaler1 / 2);
      }

      data_index += THREADS_PER_BLOCK;
    }
    __syncthreads();

    /* Calculate constant K <- u %*% p / (norm(u)^2)                          */
    /* Again, not a parallel operation, only first thread of each block used  */
    if (thread_index == DEFAULT_DATA_INDEX) {
      
      /* Store H = norm(u)^2 / 2 before calculating u %*% p */
      scaler1 /= 2;
      scaler2 = 0;
      
      /* vector u is 0 for index [0 - i], only need to multiply and sum (i + 1) onwards  */
      for (int j = (i + 1); j < matrix_arg.rdim; j++) {
        scaler2 += (shared_arr[j] * linalg_vec[j]);
      }

      /* Use shared var to store K by dividing by 2H  */
      scaler2 /= (scaler1 * 2);
    }
    __syncthreads();

    /* Calculate vector q <- p - K * u  */
    data_index = thread_index;
    while (data_index < matrix_arg.rdim) {
      
      /* ovewrite vector p in linalg_vec with vector q  */
      linalg_vec[data_index] -= (scaler2 * shared_arr[data_index]);
    }
    __syncthreads();

    /* Update tridiagonal matrix with computationally cheap but equivalent formula  */
    /* Naive formula:  A' <- P %*% A %*% P with P <- diag(n) - (u %*% t(u) / H)     */
    /* Computationally useful formula: A' <- A - (q %*% t(u)) - (u %*% t(q))        */
    /* The second formula can be calculated in place with only 2 multiplications    */
    /* Vector q is stored in linalg_vec, while vector u is stored in shared_arr     */
    data_index = grid_index;
    row_index = data_index % matrix_arg.rdim;
    col_index = data_index / matrix_arg.cdim;
    for (int j = 0; j < evals_per_thread; j++) {
      
      /* Check overflow */
      if (data_index > matrix_arg.len) break;

      /* Update tridiagonal matrix A' */
      gpu_tridiagonal[data_index] -= (linalg_vec[row_index] * shared_arr[col_index] +
                                      shared_arr[row_index] * linalg_vec[col_index]);

      data_index += grid_size;
      row_index = data_index % matrix_arg.rdim;
      col_index = data_index / matrix_arg.cdim;
    }

    /* Update accumulating matrix Q with Q' <- Q %*% P, do not explicitly create P  */
    /* in any memory as we can avoid global memory reads by instead performing      */
    /* repeated multiplications for each index of P that is recalculated, store the */
    /* final result in global evaluation memory before writing it back to gpu_Q     */
    data_index += grid_size;
    row_index = data_index % matrix_arg.rdim;
    col_index = data_index / matrix_arg.cdim;
    for (int j = 0; j < evals_per_thread; j++) {

      /* Check overflow */
      if (data_index > matrix_arg.len) break;

      /* Update global memory evaluations */
      eval_memory[data_index] = 0;
      for (int k = 0; k < matrix_arg.rdim; k++) {

        /* determine base_p for base_p - u[id1] * u[id2]  */
        // double base_p = (row_index == col_index) ? 1.0 : 0.0;
        double p = (k == col_index) ? 1.0 : 0.0;
        p -= (shared_arr[k] * shared_arr[col_index] / scaler1);
        eval_memory[data_index] += (gpu_Q[(k * matrix_arg.rdim) + row_index] * p);
      }

      data_index += grid_size;
      row_index = data_index % matrix_arg.rdim;
      col_index = data_index / matrix_arg.cdim;
    }
    grid.sync();

    /* Write the evaluated data back to gpu_Q */
    data_index = grid_index;
    for (int j = 0; j < evals_per_thread; j++) {
      gpu_Q[data_index] = eval_memory[data_index];
      data_index += grid_size;
    }
  }
}


/*
 * Top level function to sample from multivariate normal distribution 
 */

__device__ void mvrnorm_device(double* result, double* means, Rvar covar_matrix, 
                               double* gpu_Q, double* gpu_tridiagonal, double* gpu_eigvecs, 
                               double* gpu_eigvals, double* shared_arr, 
                               double* scratch_memory, double* linalg_vec, 
                               int grid_size, int grid_index, int thread_index, 
                               int evals_per_thread, cooperative_groups::grid_group grid) {
  
  /* First find eigenvectors and eigenvalues of the covariance matrix */
  
  /* Step 1 is to reduce matrix to tridiagonal form and save accumlated matrix Q  */
  householder_reduction(covar_matrix, gpu_Q, gpu_tridiagonal, shared_arr, scratch_memory, 
                        linalg_vec, grid_size, grid_index, thread_index, evals_per_thread, 
                        grid);

  
  

}


/*
 * Kernel function ran on the GPU
 */

__global__
void kernel(int grid_size, unsigned long long random_seed, double* scratch_gpu_memory,
            double* gpu_Q, double* gpu_tridiagonal, double* gpu_eigvecs, double* gpu_eigvals)
{
  /* Shared memory used for storage of evaluations or temporarily saved data, */
  /* such as the column of interest in parallel Gauss-Jordan inverse          */
  __shared__ double shared_arr[THREADS_PER_BLOCK * MAX_EVALS_PER_THREAD];
  double* temp_evals = shared_arr;

  /* Linear algebra __shared__ storage, due to hardware this is limited */
  /* Only necessary when more than one vector must be stored for linear */
  /* algebra functions such as reduction to tridiagonal form            */
  __shared__ double linalg_vec[MAX_LINALG_DIM];

  /* The indices that identify both thread index (repeated over blocks) */
  /* and the unique grid index that each thread posseses                */
  const int grid_index = blockDim.x * blockIdx.x + threadIdx.x;
  const int thread_index = threadIdx.x;

  /* Local indices used to temporarily store evaluated values before  */
  /* writing them back to the global memory of the associated Rvar    */
  int _storage_index = thread_index;
  int _storage_inc = THREADS_PER_BLOCK;
  int _eval_data_index = grid_size;
  int _guard_len = 0;

  /* Initialized the group on all threads to allow grid level synchronization */
  cooperative_groups::grid_group grid = cooperative_groups::this_grid();
  
  /* Initialize random state for RNG  */
  hiprandStateXORWOW_t* grid_state = (hiprandStateXORWOW_t*) 
                                     malloc(sizeof(hiprandStateXORWOW_t));
  hiprand_init(random_seed, grid_index, 0, grid_state);


  // [[Kernel::start]]
  // Machine generated code
  // [[Kernel::end]]
}


/*
 * Top level function called from .cpp code to start the kernel
 */

void call_device() {

  /* Copy the Rvars into __constant__ memory for faster execution in kernel */
  store_vars();

  /* Initialize and copy intermediate evaluation variables  */
  initialize_int_evals();
  store_int_evals();
  
  /* Intialize and copy iter lens into __constant__ memory for faster execution in kernel */
  initialize_iter_lens();
  store_iter_lens();

  /* Intialize and copy expr lens into __constant__ memory for faster execution in kernel */
  initialize_expr_lens();
  store_expr_lens();
  int max_evals = *(std::max_element(g_evals_per_thread, g_evals_per_thread + g_expr_count));

  /* Calculate the number of evals needed per block and raise error if this exceeds */
  /* the maximum number of evaluations per block that has been pre calculated based */
  /* on a CUDA device with at least 48kb of __shared__ memory per SM. Note that the */
  /* evals per thread in each expression will vary as the goal is to maximize       */
  /* concurrency, thus larger expressions will require more evaluations per thread. */
  hipDeviceProp_t deviceProp;
  int dev = 0;
  hipGetDeviceProperties(&deviceProp, dev);
  int grid_size = deviceProp.multiProcessorCount * BLOCKS_PER_SM * THREADS_PER_BLOCK;

  // CHECK HOW EFFICIENT 2 BLOCKS_PER_SM IS WITH SIMILAR SHARED MEMSIZE PER BLOCK
  printf("Launching %d blocks with %d threads per block\n",
         deviceProp.multiProcessorCount * BLOCKS_PER_SM, THREADS_PER_BLOCK);
  printf("Maximum concurrent evaluation of %d evals per thread\n", 
         max_evals);
  

  /* Retrieve random seed from R  */
  unsigned long long random_seed = 420;

  int linalg_dim = 1;  /* R::linalg_dim */ 

  /* Allocate background memory on GPU  */
  allocate_background_mem(max_evals * deviceProp.multiProcessorCount * BLOCKS_PER_SM * 
                          THREADS_PER_BLOCK, linalg_dim);

  /* Create argument array to be passed to kernel */
  void* args[] = {&grid_size, &random_seed, &scratch_gpu_memory, gpu_Q, gpu_tridiagonal,
                  gpu_eigvecs, gpu_eigvals};

  /* Launch kernel with cooperative groups, allowing grid wide synchronization  */
  hipLaunchCooperativeKernel((void*) kernel, deviceProp.multiProcessorCount * BLOCKS_PER_SM, 
                              THREADS_PER_BLOCK, args);

  /* Require GPU synchronize before CPU resume execution  */
  hipDeviceSynchronize();

  /* Check for any errors during kernel launch  */
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("CUDA error: %s\n", hipGetErrorString(err));
    return;
  }

  /* Clean up memory from intermediate evaluations on GPU */
  free_background_mem();
}


/*
 * Initializes the lengths of iteration loops using machine generated expressions,
 * this is called once at the start of each execution
 */

void initialize_iter_lens() {
  
  double access_mem[MAX_ITERS];
  
  /* The code below is updated by R code with expressions that are evaluated  */
  /* at each execution of the compiled commands to get the iteration length   */
  /* of each included loop                                                    */

  // [[Iter.mem::start]]
  /* Copy any memory accesses needed from GPU memory to CPU memory  */
  // [[Iter.mem::end]]


  // [[Iter.lens::start]]
  g_iter_lens[/*x*/] = /* parsed expr len */
  g_iter_count = /* R::g_loop_count */;
  // [[Iter.lens::end]]
}


/*
 * Initializes the lengths of iteration loops using machine generated expressions,
 * this is called once at the start of each execution
 */

void initialize_expr_lens() {

  /* Retrieve the grid size to allow calculation of expr specific evals per thread  */
  hipDeviceProp_t deviceProp;
  int dev = 0;
  hipGetDeviceProperties(&deviceProp, dev);
  int grid_size = THREADS_PER_BLOCK * deviceProp.multiProcessorCount * BLOCKS_PER_SM;
  int expr_len = 0;
  double access_mem[MAX_EXPRS];

  // [[Expr.mem::start]]
  /* Copy any memory accesses needed from GPU memory to CPU memory  */
  // [[Expr.mem::end]]

  // [[Expr.lens::start]]
  expr_len = /* parsed expr len */;
  g_evals_per_thread[/*x*/] = ceil((float) expr_len / grid_size);
  g_expr_count = /* R::g_expr_count */;
  // [[Expr.lens::end]]
   
}


/*
 * Initializes the lengths and dimensions of intermediate evaluation variables
 * used to store evaluations of matrix function arguments
 */

void initialize_int_evals() {
  
  double access_mem[MAX_INT_VARS];

  // [[Int.mem::start]]
  /* Copy any memory accesses needed from GPU memory to CPU memory  */
  // [[Int.mem::end]]


  /* Used to intitialize len and data fields  */
  int len = 0;  

  // [[Int.evals::start]]
  len = /* parsed expr len */
  g_int_evals[/*x*/] = {
    .data = (double*) malloc_device(sizeof(double) * len),
    .len = len,
    .rdim = /* parsed expr rdim */,
    .cdim = /* parsed expr cdim */
  };
  g_int_eval_count = /* R::g_int_eval_count */;
  // [[Int.evals::end]]

}


/*
 * Allocates global memory used for background storage
 */

void allocate_background_mem(int max_eval_size, int linalg_dim){
  scratch_gpu_memory = (double*) malloc_device(max_eval_size * sizeof(double));
  gpu_Q = (double*) malloc_device(pow(linalg_dim, 2) * sizeof(double));
  gpu_tridiagonal = (double*) malloc_device(pow(linalg_dim, 2) * sizeof(double));
  gpu_eigvecs = (double*) malloc_device(pow(linalg_dim, 2) * sizeof(double));
  gpu_Q = (double*) malloc_device(linalg_dim * sizeof(double));
}


/*
 * Frees the allocated memory associated with intermediate evaluations
 */

void free_int_evals() {
  for (int i = 0; i < g_int_eval_count; i++) {
    free_device(g_int_evals[i].data);
  }
}


/*
 * Frees all non variable memory allocated on GPU
 */

void free_background_mem() {
  free_int_evals();
  free_device(gpu_Q);
  free_device(gpu_tridiagonal);
  free_device(gpu_eigvecs);
  free_device(gpu_eigvals);
}


/*
 * Copies variable info stored in CPU memory to __constant__ GPU memory
 */

void store_vars() {
  hipError_t err = hipMemcpyToSymbol(HIP_SYMBOL(gpu_vars), g_vars, sizeof(Rvar) * g_var_count);
  if (err != hipSuccess) {
    printf("CUDA error while copying Rvars to __constant__ memory: %s\n", hipGetErrorString(err));
  }
  hipDeviceSynchronize();
}


/*
 * Copies iteration loop info stored in CPU memory to __constant__ GPU memory
 */

void store_iter_lens() {
  hipError_t err = hipMemcpyToSymbol(HIP_SYMBOL(gpu_iter_lens), g_iter_lens, sizeof(int) * g_iter_count);
  if (err != hipSuccess) {
    printf("CUDA error while copying iteration lengths to __constant__ memory: %s\n", 
           hipGetErrorString(err));
  }
  hipDeviceSynchronize();
}


/*
 * Copies expression length info stored in CPU memory to __constant__ GPU memory
 */

void store_expr_lens() {
  hipError_t err = hipMemcpyToSymbol(HIP_SYMBOL(gpu_evals_per_thread), g_evals_per_thread, 
                                       sizeof(int) * g_expr_count);
  if (err != hipSuccess) {
    printf("CUDA error while copying expression lengths to __constant__ memory: %s\n",
           hipGetErrorString(err));
  }
  hipDeviceSynchronize();
}


/*
 * Copies intermediate Rvar structures from CPU memory to __constant__ memory
 */

void store_int_evals() {
  hipError_t err = hipMemcpyToSymbol(HIP_SYMBOL(gpu_int_evals), g_int_evals, sizeof(Rvar) * g_int_eval_count);
  if (err != hipSuccess) {
    printf("CUDA error while copying intermediate evaluations to __constant__ memory: %s\n", 
           hipGetErrorString(err));
  }
  hipDeviceSynchronize();
}


