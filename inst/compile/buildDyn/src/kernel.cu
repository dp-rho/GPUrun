#include "hip/hip_runtime.h"
#include "commands.h"
#include "cuda_headers.h"

/* Memory for Rvar structures stored in __constant__ access memory for faster execution */
__constant__ Rvar gpu_vars[MAX_VARS];

/* Memory for size of loop iterations stored in __constant__ access memory for faster execution */
__constant__ int gpu_iter_lens[MAX_ITERS];

/* Memory for size of expressions stored in __constant__ access memory for faster execution */
__constant__ int gpu_evals_per_thread[MAX_EXPRS];


/* Define functions available to kernel */

/*
 * Basic addition
 */

__device__ double add(double arg1, double arg2) {
  return arg1 + arg2;
}


/*
 * Basic subtraction
 */

__device__ double sub(double arg1, double arg2) {
  return arg1 - arg2;
}


/*
 * Basic multiplication
 */

__device__ double mul(double arg1, double arg2) {
  return arg1 * arg2;
}


/*
 * Basice division
 */

__device__ double dvs(double arg1, double arg2) {
  return arg1 / arg2;
}


/*
 * Range operator, i.e., from 1:10, return 1, 2, 3, ...
 */

__device__ double range(double arg1, double arg2, int data_index) {
  int sign = (arg2 > arg1) ? 1 : -1;
  int len = std::floor(abs(arg2 - arg1) + 1);
  return arg1 + (sign * (data_index % len));
}


/*
 * Kernel function ran on the GPU
 */

__global__
void kernel(int grid_size)
{
  __shared__ double evals[THREADS_PER_BLOCK * MAX_EVALS_PER_THREAD];
  int data_index = blockDim.x * blockIdx.x + threadIdx.x;
  int thread_index = threadIdx.x;
  int _shared_mem_index = 0;
  int _eval_data_index = 0;
  cooperative_groups::grid_group grid = cooperative_groups::this_grid();

  // [[Kernel.start]]
  // Machine generated code
  // [[Kernel.end]]
}


/*
 * Top level function called from .cpp code to start the kernel
 */

void call_device() {
  /* Copy the Rvars into __constant__ memory for faster execution in kernel */
  store_vars();
  
  /* Intialize and copy the iter lens into __constant__ memory for faster execution in kernel */
  initialize_iter_lens();
  store_iter_lens();

  /* Intialize and copy the expr lens into __constant__ memory for faster execution in kernel */
  initialize_expr_lens();
  store_expr_lens();
  int max_len = *(std::max_element(g_evals_per_thread, g_evals_per_thread + g_expr_count));

  /* Calculate the number of evals needed per block and raise error if this exceeds */
  /* the maximum number of evaluations per block that has been pre calculated based */
  /* on a CUDA device with 48kb of __shared__ memory per SM.  Note that the evals   */
  /* per thread in each expression will vary as the goal is to maximize concurrency */
  /* and thus larger expressions will require more evaluations per thread.          */
  hipDeviceProp_t deviceProp;
  int dev = 0;
  hipGetDeviceProperties(&deviceProp, dev);
  int grid_size = THREADS_PER_BLOCK * deviceProp.multiProcessorCount * BLOCKS_PER_SM;
  int evals_per_thread = std::ceil((float) max_len / grid_size);
  
  if (evals_per_thread > MAX_EVALS_PER_THREAD) {
    printf("Error: Data too large for simultaneous execution on device\n");
  }
  else {
    // CHECK HOW EFFICIENT 2 BLOCKS_PER_SM IS WITH SIMILAR SHARED MEMSIZE PER BLOCK
    printf("Launching %d blocks with %d threads per block\n",
           deviceProp.multiProcessorCount * BLOCKS_PER_SM, THREADS_PER_BLOCK);
  }

  void* args[] = {&grid_size};

  hipLaunchCooperativeKernel((void*) kernel, deviceProp.multiProcessorCount * BLOCKS_PER_SM, 
                              THREADS_PER_BLOCK, args);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();
  
}


/*
 * Initializes the lengths of iteration loops using machine generated expressions,
 * this is called once at the start of each execution
 */

void initialize_iter_lens() {

  /* The code below is updated by R code with expressions that are evaluated  */
  /* at each execution of the compiled commands to get the iteration length   */
  /* of each included loop                                                    */

  // [[Iter.lens::start]]
  g_iter_lens[/*x*/] = /* parsed expr len */
  g_iter_count = /* R::g_loop_count */;
  // [[Iter.lens::end]]
}


/*
 * Initializes the lengths of iteration loops using machine generated expressions,
 * this is called once at the start of each execution
 */

void initialize_expr_lens() {

  /* Retrieve the grid size to allow calculation of expr specific evals per thread  */
  hipDeviceProp_t deviceProp;
  int dev = 0;
  hipGetDeviceProperties(&deviceProp, dev);
  int grid_size = THREADS_PER_BLOCK * deviceProp.multiProcessorCount * BLOCKS_PER_SM;
  int expr_len = 0;

  /* The code below is updated by R code with expressions that are evaluated  */
  /* at each execution of the compiled commands to get the expression length  */
  /* of each included expression                                              */

  // [[Expr.lens::start]]
  expr_len = /* parsed expr len */;
  g_evals_per_thread[/*x*/] = std::ceil((float) expr_len / grid_size);
  g_expr_count = /* R::g_expr_count */;
  // [[Expr.lens::end]]
}



/*
 * Copies variable info stored in CPU memory to __constant__ GPU memory
 */

void store_vars() {
  hipError_t err = hipMemcpyToSymbol(HIP_SYMBOL(gpu_vars), g_vars, sizeof(Rvar) * g_var_count);
  if (err != hipSuccess) {
    printf("CUDA error while copying Rvars to __constant__ memory: %s\n", hipGetErrorString(err));
  }
  hipDeviceSynchronize();
}


/*
 * Copies iteration loop info stored in CPU memory to __constant__ GPU memory
 */

void store_iter_lens() {
  hipError_t err = hipMemcpyToSymbol(HIP_SYMBOL(gpu_iter_lens), g_iter_lens, sizeof(int) * g_iter_count);
  if (err != hipSuccess) {
    printf("CUDA error while copying iteration lengths to __constant__ memory: %s\n", 
           hipGetErrorString(err));
  }
  hipDeviceSynchronize();

}


/*
 * Copies expression length info stored in CPU memory to __constant__ GPU memory
 */

void store_expr_lens() {
  hipError_t err = hipMemcpyToSymbol(HIP_SYMBOL(gpu_evals_per_thread), g_evals_per_thread, 
                                       sizeof(int) * g_expr_count);
  if (err != hipSuccess) {
    printf("CUDA error while copying expression lengths to __constant__ memory: %s\n",
           hipGetErrorString(err));
  }
  hipDeviceSynchronize();

}

